
#include <hip/hip_runtime.h>
__global__  void gSum_vec(float* v1, float* v2, int N){
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	v1[i]+=v2[i]+1.0;
}

void Sum_vec(float* v1, float* v2, float *w, int N){
  float *u1,*u2;
  
  hipMalloc((void **) &u1, N*sizeof(float));
  hipMalloc((void **) &u2, N*sizeof(float));

  hipMemcpy(u1, v1, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(u2, v2, N*sizeof(float), hipMemcpyHostToDevice);	
	
  gSum_vec<<<dim3(N/512+((N%512)?1:0)),dim3(512)>>>(u1,u2,N);
  hipDeviceSynchronize();  

  hipMemcpy(w, u1, N*sizeof(float), hipMemcpyDeviceToHost);
	
  hipFree(u1);
  hipFree(u2);
  }
